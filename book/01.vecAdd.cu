
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}


void vecAdd(float* A, float* B, float* C, int n) {

    float *A_d, *B_d, *C_d;
    int size = n * sizeof(float);

    hipMalloc((void**) &A_d, size);
    hipMalloc((void**) &B_d, size);
    hipMalloc((void**) &C_d, size);

    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);

    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {
    int N = 100000;
    std::vector<float> aaa(N);
    std::vector<float> bbb(N);
    std::vector<float> ccc(N);
    for(int i=0; i<N; ++i) {
        aaa[i] = i;
        bbb[i] = i;
    }

    vecAdd(aaa.data(), bbb.data(), ccc.data(), N);

    for(int i=0; i<N; ++i) {
        if (aaa[i] + bbb[i] != ccc[i]) {
            std::cout << "fail\n";
            return 0;
        }
    }
    std::cout << "success\n";
    return 0;
}
